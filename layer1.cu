#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void warm_up_gpu()
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float ia, ib;
	ia = ib = 0.0f;
	ib += ia + tid;
}

__global__ void layer1_w(int *p, int *q, int *w, int numNode)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid < (numNode+1) * (numNode+2) / 2)
	{
		int i = 1;
		for(int idx = (numNode+1); idx>0; idx--)
		{
			if(tid - idx < 0)
				break;
			
			tid -= idx;
			i++;
		}
		int j = i + tid - 1;

		for(int k=i-1; k<=j; k++)
		{
			w[i * (numNode+1) + j] += q[k];
		}
		for(int k=i; k<=j; k++)
		{
			w[i * (numNode+1) + j] += p[k];
		}
	}

	return;
}

__global__ void layer1_s(int *q, int *w, int *s, int k, int numNode)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid >= (numNode-k))
		return;

	int i = tid + 1;
	if(k == -1)
		s[i * (numNode+1) + i + k] = q[i + k];
	else
	{
		int r = i;
		s[i * (numNode+1) + i + k] = s[i * (numNode+1) + r - 1] \
									 + s[(r+1) * (numNode+1) + i + k] \
									 + w[i * (numNode+1) + i + k];
		for(r=i; r<=i+k; r++)
		{
			if(s[i*(numNode+1)+r-1] + s[(r+1)*(numNode+1)+i+k] + w[i*(numNode+1)+i+k] < s[i*(numNode+1)+i+k])
			  s[i*(numNode+1)+i+k] = s[i*(numNode+1)+r-1] + s[(r+1)*(numNode+1)+i+k] + w[i*(numNode+1)+i+k];
		}
	}
	
	return;
}

//layer1 front end host function
void layer1FE(int *p, int *q, int *w, int *s, int numNode)
{
	int block_size = 32;
	int grid_size = ceil( (double)(numNode+1) * (double)(numNode+2) / 2.0 / (double)block_size );

	warm_up_gpu<<<grid_size, block_size>>>();

	//compute w
	int *w_d;
	int *p_d;
	int *q_d;
	hipMalloc(&w_d, (numNode+2) * (numNode+1) * sizeof(int));
	hipMalloc(&p_d, (numNode+1) * sizeof(int));
	hipMalloc(&q_d, (numNode+1) * sizeof(int));
	hipMemcpy(p_d, p, (numNode+1) * sizeof(int), hipMemcpyDefault);
	hipMemcpy(q_d, q, (numNode+1) * sizeof(int), hipMemcpyDefault);

	layer1_w<<<grid_size, block_size>>>(p_d, q_d, w_d, numNode);

	hipDeviceSynchronize();

	hipMemcpy(w, w_d, (numNode+2) * (numNode+1) * sizeof(int), hipMemcpyDefault);

	//compute s
	int *s_d;
	hipMalloc(&s_d, (numNode+2) * (numNode+1) * sizeof(int));

	for(int k=-1; k<=(numNode-1); k++)
	{
		grid_size = ceil( (double)(numNode-k) / block_size );
		layer1_s<<<grid_size, block_size>>>(q_d, w_d, s_d, k, numNode);
		hipDeviceSynchronize();
	}

	hipMemcpy(s, s_d, (numNode+2) * (numNode+1) * sizeof(int), hipMemcpyDefault);

	hipFree(w_d);
	hipFree(p_d);
	hipFree(q_d);
	hipFree(s_d);
}
