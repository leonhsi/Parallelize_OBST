#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void warm_up()
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float ia, ib;
	ia = ib = 0.0f;
	ib += ia + tid;
}

__global__ void compute_w(int *p, int *q, int *w, int numNode)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid < (numNode+1) * (numNode+2) / 2)
	{
		int i = 1;
		for(int idx = (numNode+1); idx>0; idx--)
		{
			if(tid - idx < 0)
				break;

			tid -= idx;
			i++;
		}
		int j = i + tid - 1;

		for(int k=i-1; k<=j; k++)
		{
			w[i * (numNode+1) + j] += q[k];
		}
		for(int k=i; k<=j; k++)
		{
			w[i * (numNode+1) + j] += p[k];
		}
	}
	return;
}

__global__ void compute_min(int i, int k, int *s, int *w, int *min, int numNode)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	int r = tid + i;

	min[tid] = s[i*(numNode+1)+r-1] + s[(r+1)*(numNode+1)+i+k] + w[i*(numNode+1)+i+k];
}

__global__ void dp_s(int *q, int *w, int *s, int k, int numNode)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int i = tid + 1;
	if(k == -1)
	{
		s[i * (numNode+1) + i + k] = q[i + k];
	}
	else
	{
		int *min = (int *)malloc((k+1) * sizeof(int));

		compute_min<<<1, k+1>>>(i, k, s, w, min, numNode);

		hipDeviceSynchronize();

		int tmp = INT_MAX;
		for(int idx=0; idx<(k+1); idx++)
		{
			if(min[idx] < tmp)
			{
				tmp = min[idx];
			}
		}
		s[i * (numNode+1) + i + k] = tmp;
	}
	return;
}

//dynamic parallelism front end host function
void dpFE(int *p, int *q, int *w, int *s, int numNode)
{
	int block_size = 32;
	int grid_size = ceil((double)(numNode+1) * (double)(numNode+2) / 2.0 / (double)block_size );
	
	warm_up<<<grid_size, block_size>>>();

	//compute w
	int *w_d;
	int *p_d;
	int *q_d;
	hipMalloc(&w_d, (numNode+2) * (numNode+1) * sizeof(int));
	hipMalloc(&p_d, (numNode+1) * sizeof(int));
	hipMalloc(&q_d, (numNode+1) * sizeof(int));
	hipMemcpy(p_d, p, (numNode+1) * sizeof(int), hipMemcpyDefault);
	hipMemcpy(q_d, q, (numNode+1) * sizeof(int), hipMemcpyDefault);

	compute_w<<<grid_size, block_size>>>(p_d, q_d, w_d, numNode);

	hipDeviceSynchronize();

	hipMemcpy(w, w_d, (numNode+2) * (numNode+1) * sizeof(int), hipMemcpyDefault);

	//compute s
	int *s_d;
	hipMalloc(&s_d, (numNode+2) * (numNode+1) * sizeof(int));

	for(int k=-1; k<=(numNode-1); k++)
	{
		grid_size = ceil( (double)(numNode-k) / block_size );
		dp_s<<<(numNode-k), 1>>>(q_d, w_d, s_d, k, numNode);
		hipDeviceSynchronize();
	}

	hipMemcpy(s, s_d, (numNode+2) * (numNode+1) * sizeof(int), hipMemcpyDefault);

	hipFree(w_d);
	hipFree(p_d);
	hipFree(q_d);
	hipFree(s_d);
}
